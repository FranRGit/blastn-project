#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <string>
#include <unordered_map>
#include <fstream>
#include <chrono>
#include <omp.h>
#include <hip/hip_runtime.h>

using namespace std;
using namespace chrono;



// Definición de la estructura Seed
struct Seed {
    int pos;      // Posición del W-mero en la secuencia
    int score;    // Puntaje 
    int idxSeq;   // Identificador de la secuencia (índice de la secuencia en el archivo)
};

// 1- Entrada: Query y Secuencias BD 
void leer_fasta(const string& filename, vector<string>& names, vector<string>& sequences) {
    ifstream file(filename);
    if (!file) {
        cerr << "Error al abrir el archivo " << filename << endl;
        exit(1);
    }

    string line, sequence;
    string header;
    while (getline(file, line)) {
        if (line.empty()) continue;

        if (line[0] == '>') {
            if (!header.empty()) {
                sequences.push_back(sequence);
                sequence.clear();
            }
            names.push_back(line.substr(1));
            header = line;
        }
        else {
            sequence += line;
        }
    }

    if (!header.empty()) {
        sequences.push_back(sequence);
    }

    file.close();
}


//<=============BLAST SEQUENTIAL===============>
// 1- División de palabras - Creación de Wmers
void genrateWmers_sequential(vector <string>& sequences, vector <string>& queryWmers, int W) {
    for (int idxSeq = 0; idxSeq < sequences.size(); idxSeq++) {
        if (idxSeq == 0) {
            for (int i = 0; i <= sequences[idxSeq].size() - W; i++) {
                queryWmers.push_back(sequences[idxSeq].substr(i, W));
            }
        }
    }
}

//2-  Extensión de coincidencias
Seed extendSeed(const string& query, const string& sequence, int startQuery, int startDb, int X) {
    int score = 0, i = 0;
    while (startQuery + i < query.size() && startDb + i < sequence.size()) {
        if (query[startQuery + i] == sequence[startDb + i]) {
            score++;
        }
        else {
            score--;
        }
        if (score < X) break;
        i++;
    }
    return { startQuery, score };
}

//3- BlastN - Secuencial
Seed blastnSequential(vector<string>& dbSequences, int W, int X) {
    vector<string> queryWmers;

    genrateWmers_sequential(dbSequences, queryWmers, W);

    Seed best_seed = { 0, 0, 0 };

    for (int i = 0; i < queryWmers.size(); i++) {
        for (int idxSeq = 1; idxSeq < dbSequences.size(); idxSeq++) { // Excluyendo la consulta (dbSequences[0])
            for (int j = 0; j <= dbSequences[idxSeq].size() - W; j++) {
                if (queryWmers[i] == dbSequences[idxSeq].substr(j, W)) {
                    Seed seed = extendSeed(dbSequences[0], dbSequences[idxSeq], i, j, X);
                    if (seed.score > best_seed.score) {
                        best_seed = { seed.pos, seed.score, idxSeq };
                    }
                }
            }
        }
    }

    return best_seed;
}

//<=============BLAST PARALELO===============>
__device__ int computeScore(const char* query, const char* sequence, int startQuery, int startDb, int querySize, int sequenceSize, int X) {
    int score = 0, i = 0;
    while (startQuery + i < querySize && startDb + i < sequenceSize) {
        if (query[startQuery + i] == sequence[startDb + i]) {
            score += 2; 
        }
        else {
            score -= 1; 
        }
        if (score < X) break;
        i++;
    }
    return score;
}

__device__ bool strncmpCUDA(const char* str1, const char* str2, int n) {
    for (int i = 0; i < n; i++) {
        if (str1[i] != str2[i]) {
            return false;
        }
    }
    return true;
}

__global__ void blastnKernel(const char* query, const char* dbSequences, int* sequenceOffsets, int* sequenceLengths, int numSequences, int W, int X, Seed* bestSeed) {
    int idxSeq = blockIdx.x;
    int threadId = threadIdx.x;
    if (idxSeq == 0) return; // Evitar la consulta misma

    __shared__ Seed localBestSeed;
    if (threadId == 0) {
        localBestSeed = { 0, 0, idxSeq };
    }
    __syncthreads();

    int seqStart = sequenceOffsets[idxSeq];
    int seqLength = sequenceLengths[idxSeq];
    int queryLength = sequenceLengths[0];

    for (int j = threadId; j <= seqLength - W; j += blockDim.x) {
        for (int i = 0; i <= queryLength - W; i++) {
            if (strncmpCUDA(&query[i], &dbSequences[seqStart + j], W)) {
                int score = computeScore(query, &dbSequences[seqStart], i, j, queryLength, seqLength, X);
                if (score > localBestSeed.score) {
                    localBestSeed = { i, score, idxSeq };
                }
            }
        }
    }

    __syncthreads();
    if (threadId == 0) {
        atomicMax(&bestSeed->score, localBestSeed.score);
        if (bestSeed->score == localBestSeed.score) {
            bestSeed->pos = localBestSeed.pos;
            bestSeed->idxSeq = localBestSeed.idxSeq;
        }
    }
}

Seed blastnCUDA(const vector<string>& dbSequences, int W, int X) {
    char* d_dbSequences;
    int* d_sequenceOffsets;
    int* d_sequenceLengths;
    Seed* d_bestSeed;
    Seed bestSeed = { 0, 0, 0 };

    int numSequences = dbSequences.size();
    vector<int> sequenceOffsets(numSequences);
    vector<int> sequenceLengths(numSequences);
    int totalSize = 0;

    for (int i = 0; i < numSequences; i++) {
        sequenceOffsets[i] = totalSize;
        sequenceLengths[i] = dbSequences[i].size();
        totalSize += dbSequences[i].size();
    }

    hipMalloc(&d_dbSequences, totalSize * sizeof(char));
    hipMalloc(&d_sequenceOffsets, numSequences * sizeof(int));
    hipMalloc(&d_sequenceLengths, numSequences * sizeof(int));
    hipMalloc(&d_bestSeed, sizeof(Seed));

    hipMemcpy(d_sequenceOffsets, sequenceOffsets.data(), numSequences * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sequenceLengths, sequenceLengths.data(), numSequences * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bestSeed, &bestSeed, sizeof(Seed), hipMemcpyHostToDevice);

    string concatenatedSequences;
    for (const auto& seq : dbSequences) {
        concatenatedSequences += seq;
    }
    hipMemcpy(d_dbSequences, concatenatedSequences.c_str(), totalSize * sizeof(char), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    blastnKernel << <numSequences, threadsPerBlock >> > (d_dbSequences, d_dbSequences, d_sequenceOffsets, d_sequenceLengths, numSequences, W, X, d_bestSeed);

    hipMemcpy(&bestSeed, d_bestSeed, sizeof(Seed), hipMemcpyDeviceToHost);

    hipFree(d_dbSequences);
    hipFree(d_sequenceOffsets);
    hipFree(d_sequenceLengths);
    hipFree(d_bestSeed);

    return bestSeed;
}

void printResult(const string& method, int index, const vector<string>& dbHeader) {
    if (index != -1) {
        cout << "Mejor coincidencia (" << method << "): " << dbHeader[index] << endl;
    }
    else {
        cout << "No se encontró coincidencia en la versión " << method << "." << endl;
    }
}


int main() {
    vector<string> dbHeader, dbSequences, queryHeader;
    string query_file = "";
    string database_file = "";
    int W = 11;  // Longitud del W-meros
    int X = -3; //Umbral

    leer_fasta(query_file, queryHeader, dbSequences);
    leer_fasta(database_file, dbHeader, dbSequences);

    if (dbSequences.empty()) {
        cerr << "Error: No se encontraron secuencias en los archivos FASTA" << endl;
        return 1;
    }

    string queryName = queryHeader[0];


    auto start_seq = high_resolution_clock::now();
    Seed resultSeq = blastnSequential(dbSequences, W, X);
    auto end_seq = high_resolution_clock::now();
    double tiempo_secuencial = duration<double>(end_seq - start_seq).count();
   
 
    
    auto start_par = high_resolution_clock::now();
    Seed resultPar = blastnCUDA(dbSequences, W, X);
    auto end_par = high_resolution_clock::now();
    double tiempo_paralelo = duration<double>(end_par - start_par).count();




    cout << "Tiempo Secuencial: " << tiempo_secuencial << " segundos" << endl;
    cout << "Tiempo Paralelo: " << tiempo_paralelo << " segundos" << endl;
    cout << "Speedup: " << tiempo_secuencial / tiempo_paralelo << "x" << endl;

    cout << "Query: " << queryName << endl;
    printResult("Secuencial", resultSeq.idxSeq - 1, dbHeader);
    printResult("Paralelo", resultPar.idxSeq - 1, dbHeader);

}

